#include "hip/hip_runtime.h"
#define USE_LDG
#include "diffusion_cuda_shared.h"
#include "common/cuda_util.h"

namespace diffusion {
namespace cuda_shared4 {

#define GET(x) (x)

#define diffusion_backward()                                            \
  do {                                                                  \
    sb[ps] = s2;                                                        \
    __syncthreads();                                                    \
    f2[p-xy] = cc * s2                                                  \
        + cw * sb[ps+sb_w] + ce * sb[ps+sb_e]                           \
        + cs * sb[ps+sb_s] + cn * sb[ps+sb_n] + cb*s1 + ct*s3;          \
  } while (0)

// Temporal blocking
// no z blocking
__global__ void kernel3d(F1_DECL f1, F2_DECL f2,
                         int nx, int ny, int nz,
                         REAL ce, REAL cw, REAL cn, REAL cs,
                         REAL ct, REAL cb, REAL cc) {
  // Two halos at each side of x direction, and one halo at each side
  // of y direction
  // (BLOCK_X+4) * (BLOCK_Y+2)
  extern __shared__ REAL sb[];
  // x dimension of the shared memory
  const int sbx = BLOCK_X+4;
  const int tidx = threadIdx.x % BLOCK_X;
  const int tidy = threadIdx.x / BLOCK_X - 1;
  int i = BLOCK_X * blockIdx.x + tidx;
  int j = BLOCK_Y * blockIdx.y + tidy;
  j = (j < 0) ? 0 : j;      // max(j, 0)
  j = (j == ny) ? ny - 1 : j; // min(j, ny-1)

  int xy = nx * ny;
  const int block_z = nz / gridDim.z;
  int k = block_z * blockIdx.z;
  const int k_end = k + block_z;
  // global offset
  int p = OFFSET3D(i, j, k, nx, ny);
  // shared offset
  int ps = OFFSET2D(tidx+2, tidy+1, sbx);

  if (tidy == -1) {
    // threads in the south halo
    int s = (j == 0)        ? 0 : -nx;

    float t2 = GET(f1[p]);
    float t1 = (k == 0) ? t2 : GET(f1[p-xy]);
    float t3 = (k < nz-1) ? GET(f1[p+xy]) : t2;
    sb[ps] = t2;
    __syncthreads();
    float s2, s3;            
    s3 = cc * t2
        + cw * sb[ps-1] + ce * sb[ps+1]
        + cs * GET(f1[p+s])
        + cn * sb[ps+sbx] + cb*t1 + ct*t3;
    p += xy;
    __syncthreads();

    for (k = 1; k < k_end; ++k) {
      SHIFT3(t1, t2, t3);
      t3 = (k < nz-1) ? GET(f1[p+xy]) : t3;      
      sb[ps] = t2;

      s2 = s3;
      __syncthreads();
      
      s3 = cc * t2
          + cw * sb[ps-1] + ce * sb[ps+1]
          + cs * GET(f1[p+s])
          + cn * sb[ps+sbx] + cb*t1 + ct*t3;
      __syncthreads();
      sb[ps] = s2;
      __syncthreads();
      __syncthreads();       
      p += xy;      
    }

    s2 = s3;
    sb[ps] = s2;
    __syncthreads();    
  } else if (tidy == BLOCK_Y) {
    // threads in the north halo
    int n = (j == ny-1)     ? 0 : nx;

    float t2 = GET(f1[p]);
    float t1 = (k == 0) ? t2 : GET(f1[p-xy]);
    float t3 = (k < nz-1) ? GET(f1[p+xy]) : t2;
    sb[ps] = t2;
    __syncthreads();
    float s2, s3;      
    s2 = s3 = cc * t2
        + cw * sb[ps-1] + ce * sb[ps+1]
        + cs * sb[ps-sbx] + cn * GET(f1[p+n]) + cb*t1 + ct*t3;
    p += xy;
    __syncthreads();
    for (k = 1; k < k_end; ++k) {
      SHIFT3(t1, t2, t3);
      t3 = (k < nz-1) ? GET(f1[p+xy]) : t3;      
      sb[ps] = t2;
      s2 = s3;
      __syncthreads();
      
      s3 = cc * t2
          + cw * sb[ps-1] + ce * sb[ps+1]
          + cs * sb[ps-sbx] + cn * GET(f1[p+n]) + cb*t1 + ct*t3;
      __syncthreads();
      sb[ps] = s2;
      __syncthreads();
      __syncthreads();      
      p += xy;      
    }
    s2 = s3;
    sb[ps] = s2;
    __syncthreads();
  } else if (tidy >= 0 && tidy < BLOCK_Y) {
    int sb_s = (j == 0)    ? 0: -sbx;
    int sb_n = (j == ny-1) ? 0:  sbx; 
    int sb_w = (i == 0)    ? 0: -1;
    int sb_e = (i == nx-1) ? 0:  1;

    float t2 = GET(f1[p]);
    float t1 = (k == 0) ? t2 : GET(f1[p-xy]);
    float t3 = (k < nz-1) ? GET(f1[p+xy]) : t2;
    sb[ps] = t2;
    __syncthreads();
    float s1, s2, s3;
    s2 = s3 = cc * t2
        + cw * sb[ps-1] + ce * sb[ps+1]
        + cs * sb[ps-sbx]+ cn * sb[ps+sbx]
        + cb * t1 + ct * t3;
    p += xy;
    __syncthreads();
    for (k = 1; k < k_end; ++k) {          
      SHIFT3(t1, t2, t3);
      t3 = (k < nz-1) ? GET(f1[p+xy]) : t3;      
      sb[ps] = t2;
      SHIFT3(s1, s2, s3);      
      __syncthreads();
    
      s3 = cc * t2
          + cw * sb[ps-1] + ce * sb[ps+1]
          + cs * sb[ps-sbx]+ cn * sb[ps+sbx]
          + cb * t1 + ct * t3;
      __syncthreads();
      diffusion_backward();
      __syncthreads();
      p += xy;            
    }
    SHIFT3(s1, s2, s3);
    diffusion_backward();
  } else {
    // horizontal halo
    int xoffset = (tidx & 1) + ((tidx & 2) >> 1) * (BLOCK_X + 2);
    int yoffset = tidx >> 2;
    yoffset = (yoffset >= (BLOCK_Y + 2)) ? BLOCK_Y+1 : yoffset;
    i = BLOCK_X * blockIdx.x - 2 + xoffset;
    i = (i < 0) ? 0 : i;
    i = (i >= nx) ? nx - 1 : i;
    j = BLOCK_Y * blockIdx.y -1 + yoffset;
    j = (j < 0) ? 0 : j;      // max(j, 0)
    j = (j >= ny) ? ny - 1 : j; // min(j, ny-1)

    int s = (yoffset == 0)  ? 0 : -sbx;
    int n = (yoffset == BLOCK_Y+1) ? 0 : sbx;
    int w = (xoffset == 0) ? 0 : -1;
    int e = (xoffset == sbx-1) ? 0 : 1;
    
    p = i + j * nx + k * xy;
    ps = xoffset + yoffset * sbx;
    
    float t2 = GET(f1[p]);
    float t1 = (k == 0) ? t2 : GET(f1[p-xy]);
    float t3 = (k < nz-1) ? GET(f1[p+xy]) : t2;
    sb[ps] = t2;
    __syncthreads();
    float s2, s3;
    s2 = s3 = cc * t2
        + cw * sb[ps+w] + ce * sb[ps+e]
        + cs * sb[ps+s] + cn * sb[ps+n]
        + cb*t1 + ct*t3;
    __syncthreads();
    p += xy;      

    for (k = 1; k < k_end-1; ++k) {
      SHIFT3(t1, t2, t3);
      t3 = GET(f1[p+xy]);
      sb[ps] = t2;
      s2 = s3;
      __syncthreads();
      s3 = cc * t2
          + cw * sb[ps+w] + ce * sb[ps+e]
          + cs * sb[ps+s] + cn * sb[ps+n]
          + cb*t1 + ct*t3;
      __syncthreads();
      sb[ps] = s2;
      __syncthreads();
      __syncthreads();      
      p += xy;      
    }

    SHIFT3(t1, t2, t3);
    t3 = (k < nz-1) ? GET(f1[p+xy]) : t3;      
    sb[ps] = t2;
    s2 = s3;
    __syncthreads();
    s3 = cc * t2
        + cw * sb[ps+w] + ce * sb[ps+e]
        + cs * sb[ps+s] + cn * sb[ps+n]
        + cb*t1 + ct*t3;
    __syncthreads();
    sb[ps] = s2;
    __syncthreads();
    __syncthreads();      
    p += xy;      
    
    s2 = s3;
    sb[ps] = s2;
    __syncthreads();
  }
  return;
}

} // namespace cuda_shared4

void DiffusionCUDAShared4::RunKernel(int count) {
  size_t s = sizeof(REAL) * nx_ * ny_ * nz_;  
  FORCE_CHECK_CUDA(hipMemcpy(f1_d_, f1_, s, hipMemcpyHostToDevice));
  assert(count % 2 == 0);
  //dim3 block_dim(BLOCK_X * BLOCK_Y + 32); // + 1 warp
  dim3 block_dim(BLOCK_X * (BLOCK_Y+2) + 32);
  dim3 grid_dim(nx_ / BLOCK_X, ny_ / BLOCK_Y, 1);
  CHECK_CUDA(hipEventRecord(ev1_));
  for (int i = 0; i < count; i+=2) {
    cuda_shared4::kernel3d<<<grid_dim, block_dim,
        (BLOCK_X+4)*(BLOCK_Y+2)*sizeof(float)>>>
        (f1_d_, f2_d_, nx_, ny_, nz_, ce_, cw_, cn_, cs_, ct_, cb_, cc_);
    REAL *t = f1_d_;
    f1_d_ = f2_d_;
    f2_d_ = t;
  }
  CHECK_CUDA(hipEventRecord(ev2_));
  FORCE_CHECK_CUDA(hipMemcpy(f1_, f1_d_, s, hipMemcpyDeviceToHost));
  return;
}

void DiffusionCUDAShared4::Setup() {
  DiffusionCUDA::Setup();
  FORCE_CHECK_CUDA(hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuda_shared4::kernel3d),
                                          hipFuncCachePreferShared));
}

} // namespace diffusion

