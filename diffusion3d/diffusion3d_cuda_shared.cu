#include "hip/hip_runtime.h"
#include "diffusion3d_cuda.h"

#define CUDA_SAFE_CALL(c)                       \
  do {                                          \
    assert(c == hipSuccess);                   \
  } while (0)

namespace diffusion3d {

__global__ void diffusion_kernel_shared(F1_DECL, REAL *__restrict f2,
                                        int nx, int ny, int nz,
                                        REAL ce, REAL cw, REAL cn, REAL cs,
                                        REAL ct, REAL cb, REAL cc) {
  const int tid_x = threadIdx.x;
  const int tid_y = threadIdx.y;
  const int i = blockDim.x * blockIdx.x + tid_x;
  const int j = blockDim.y * blockIdx.y + tid_y;
  const int xy = nx * ny;
  __shared__ REAL sb[BLOCK_X * BLOCK_Y];
  const int block_z = nz / gridDim.z;
  int k = block_z * blockIdx.z;
  const int k_end = k + block_z;
  int c = i + j * nx + k * xy;
  const int c1 = tid_x + tid_y * blockDim.x;
  REAL t1, t2, t3;
  t3 = f1[c];
  t2 = (k == 0) ? t3 : f1[c-xy];
  int w = (i == 0)        ? c1 : c1 - 1;
  int e = (i == nx-1)     ? c1 : c1 + 1;
  int n = (j == 0)        ? c1 : c1 - blockDim.x;
  int s = (j == ny-1)     ? c1 : c1 + blockDim.x;
  int bw = tid_x == 0 && i != 0;
  int be = tid_x == blockDim.x-1 && i != nx - 1;
  int bn = tid_y == 0 && j != 0;
  int bs = tid_y == blockDim.y-1 && j != ny - 1;

#pragma unroll
  for (; k < k_end-1; ++k) {
    t1 = t2;
    t2 = t3;
    sb[c1] = t2;    
    t3 = f1[c+xy];
    REAL t = cc * t2 + cb * t1 + ct * t3;    
    __syncthreads();
    t += cw * (bw ? f1[c-1] : sb[w]);
    t += ce * (be ? f1[c+1] : sb[e]);
    t += cs * (bs ? f1[c+nx] : sb[s]);
    t += cn * (bn ? f1[c-nx] : sb[n]);
    f2[c] = t;
    c += xy;
    __syncthreads();
  }
  t1 = t2;
  t2 = t3;
  sb[c1] = t2;    
  t3 = (k < nz-1) ? f1[c+xy] : t3;
  REAL t = cc * t2 + cb * t1 + ct * t3;    
  __syncthreads();
  t += cw * (bw ? f1[c-1] : sb[w]);
  t += ce * (be ? f1[c+1] : sb[e]);
  t += cs * (bs ? f1[c+nx] : sb[s]);
  t += cn * (bn ? f1[c-nx] : sb[n]);
  f2[c] = t;
  return;
}

void Diffusion3DCUDAShared::InitializeBenchmark() {
  Diffusion3DCUDA::InitializeBenchmark();
  CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(diffusion_kernel_shared),
                                        hipFuncCachePreferShared));
}

void Diffusion3DCUDAShared::RunKernel(int count) {
  int flag = 0;
  assert(nx_ % block_x_ == 0);
  assert(ny_ % block_y_ == 0);
  assert(nx_ / block_x_ > 0);
  assert(ny_ / block_y_ > 0);
  
  size_t s = sizeof(REAL) * nx_ * ny_ * nz_;  
  CUDA_SAFE_CALL(hipMemcpy(f1_d_, f1_, s, hipMemcpyHostToDevice));

  dim3 block_dim(block_x_, block_y_, 1);
  dim3 grid_dim(nx_ / block_x_, ny_ / block_y_, grid_z_);

  #pragma omp parallel num_threads(2) shared(flag)
  {
    if (omp_get_thread_num() == 0)
    {
      power = GetPowerGPU(&flag, 0);
    }
    else
    {
      #pragma omp barrier
      CUDA_SAFE_CALL(hipEventRecord(ev1_));
      for (int i = 0; i < count; ++i) {
        diffusion_kernel_shared<<<grid_dim, block_dim>>>
            (f1_d_, f2_d_, nx_, ny_, nz_, ce_, cw_, cn_, cs_, ct_, cb_, cc_);
        REAL *t = f1_d_;
        f1_d_ = f2_d_;
        f2_d_ = t;
      }
      CUDA_SAFE_CALL(hipEventRecord(ev2_));
      CUDA_SAFE_CALL(hipDeviceSynchronize());
      flag = 1;
    }
  }

  CUDA_SAFE_CALL(hipMemcpy(f1_, f1_d_, s, hipMemcpyDeviceToHost));
  return;
}

}

